
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cfloat>

#define TILE_WIDTH 32
#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};
float checkCorrectFloat(float * a1, float* a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)	
		err += abs(a1[i] - a2[i]);
	err /= n;
	return err;
}
void printDeviceInfo()
{
	hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("****************************\n\n");

}
// ===================== Transpose =====================
__global__ void transposeKernelV1(float *input,  float *output, int numCols, int numRows)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < numRows && col < numCols)
    {
        output[col * numRows + row] = input[row * numCols + col];
    }
}

__global__ void transposeKernelV2(float *in,  float *out, int numCols, int numRows) {
    __shared__ float s_blkData[32][32];

    int iR = blockIdx.y * blockDim.y + threadIdx.y;
    int iC = blockIdx.x * blockDim.x + threadIdx.x;
    if (iR < numRows && iC < numCols) s_blkData[threadIdx.x][threadIdx.y] = in[iR * numCols + iC];
    __syncthreads();

    int oC = blockIdx.y * blockDim.y + threadIdx.x;
    int oR = blockIdx.x * blockDim.x + threadIdx.y;
    if (oR < numCols && oC < numRows) out[oR * numRows + oC] = s_blkData[threadIdx.y][threadIdx.x];
}

__global__ void transposeKernelV3(float *in,  float *out, int numCols, int numRows) {
    __shared__ float s_blkData[32][33];

    int iR = blockIdx.y * blockDim.y + threadIdx.y;
    int iC = blockIdx.x * blockDim.x + threadIdx.x;
    if (iR < numRows && iC < numCols) s_blkData[threadIdx.x][threadIdx.y] = in[iR * numCols + iC];
    __syncthreads();

    int oC = blockIdx.y * blockDim.y + threadIdx.x;
    int oR = blockIdx.x * blockDim.x + threadIdx.y;
    if (oR < numCols && oC < numRows) out[oR * numRows + oC] = s_blkData[threadIdx.y][threadIdx.x];
}

void transposeHost(float* in, float* out, int length, int width) {
    for(int i = 0; i < width; i++) {
        for (int j = 0; j < length; j++) {
            out[j * length + i] = in[i * width + j];
        }
    }
}

void transpose(float * in, float * out, int length, int width,
    int verDevice = 0, dim3 blockSize = dim3(1))
{
    GpuTimer timer;
    timer.Start();
	if (verDevice == 0)
	{
		transposeHost(in, out, length, width);
	}
	else if (verDevice == 1) // Use device
	{
		// Allocate device memories
		float * d_in, * d_out;
		dim3 gridSize((width - 1) / blockSize.x + 1, 
					(length - 1) / blockSize.y + 1); // TODO: Compute gridSize from n and blockSize
		
		// TODO: Allocate device memories
        CHECK(hipMalloc(&d_in, width * length * sizeof(float)));
        CHECK(hipMalloc(&d_out, width * length * sizeof(float)));
		// TODO: Copy data to device memories
        CHECK(hipMemcpy(d_in, in, width * length * sizeof(float), hipMemcpyHostToDevice));
		

		// Call kernel
		transposeKernelV1<<<gridSize, blockSize>>>(d_in, d_out, length, width);

		hipDeviceSynchronize();

		CHECK(hipGetLastError());
		
		// TODO: Copy result from device memories
        CHECK(hipMemcpy(out, d_out, width * length * sizeof(float), hipMemcpyDeviceToHost));
		// TODO: Free device memories
        CHECK(hipFree(d_in));
        CHECK(hipFree(d_out));
		// Print info
		printf("Kernel version 1, Grid size: %d, block size: %d\n", gridSize.x, blockSize.x);
	}
    else if (verDevice == 2) {
        // Allocate device memories
		float * d_in, * d_out;
		dim3 gridSize((width - 1) / blockSize.x + 1, 
					(length - 1) / blockSize.y + 1); // TODO: Compute gridSize from n and blockSize
		
		// TODO: Allocate device memories
        CHECK(hipMalloc(&d_in, width * length * sizeof(float)));
        CHECK(hipMalloc(&d_out, width * length * sizeof(float)));
		// TODO: Copy data to device memories
        CHECK(hipMemcpy(d_in, in, width * length * sizeof(float), hipMemcpyHostToDevice));

		// Call kernel
		transposeKernelV2<<<gridSize, blockSize>>>(d_in, d_out, length, width);

		hipDeviceSynchronize();

		CHECK(hipGetLastError());
		
		// TODO: Copy result from device memories
        CHECK(hipMemcpy(out, d_out, width * length * sizeof(float), hipMemcpyDeviceToHost));
		// TODO: Free device memories
        CHECK(hipFree(d_in));
        CHECK(hipFree(d_out));
		// Print info
		printf("Kernel version 2, Grid size: %d, block size: %d\n", gridSize.x, blockSize.x);
    }
    else {
        // Allocate device memories
		float * d_in, * d_out;
		dim3 gridSize((width - 1) / blockSize.x + 1, 
					(length - 1) / blockSize.y + 1); // TODO: Compute gridSize from n and blockSize
		
		// TODO: Allocate device memories
        CHECK(hipMalloc(&d_in, width * length * sizeof(float)));
        CHECK(hipMalloc(&d_out, width * length * sizeof(float)));
		// TODO: Copy data to device memories
        CHECK(hipMemcpy(d_in, in, width * length * sizeof(float), hipMemcpyHostToDevice));

		// Call kernel
		transposeKernelV3<<<gridSize, blockSize>>>(d_in, d_out, length, width);

		hipDeviceSynchronize();

		CHECK(hipGetLastError());
		
		// TODO: Copy result from device memories
        CHECK(hipMemcpy(out, d_out, width * length * sizeof(float), hipMemcpyDeviceToHost));
		// TODO: Free device memories
        CHECK(hipFree(d_in));
        CHECK(hipFree(d_out));
		// Print info
		printf("Kernel version 3, Grid size: %d, block size: %d\n", gridSize.x, blockSize.x);
    }

    timer.Stop();
    float time = timer.Elapsed();
    if (verDevice == 0) {
        printf("Host time : %f ms\n\n", time);
    }
    else if (verDevice == 1) {
        printf("Kernel version 1 time : %f ms\n", time);
    }
    else if (verDevice == 2) {
        printf("Kernel version 2 time : %f ms\n", time);
    }
    else {
        printf("Kernel version 3 time : %f ms\n", time);
    }
}

void checkTranspose(int argc, char ** argv){
    printf("**************** Transpose Matrix ****************\n");

    //Declare variables
    int length = (1 << 12);
    int width = (1 << 12);

    float * in = (float *) malloc(width * length * sizeof(float));
    float * outHost = (float *) malloc(width * length * sizeof(float));
    float * outDevice1 = (float *) malloc(width * length * sizeof(float));
    float * outDevice2 = (float *) malloc(width * length * sizeof(float));
    float * outDevice3 = (float *) malloc(width * length * sizeof(float));
    for (int i = 0; i < width * length; i++)
    {
        in[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
        // Get randon negative value
        if (rand() % 2 == 0)
            in[i] = -in[i];
    }
    transpose(in, outHost, length, width);

    dim3 blockSize(32, 32); // Default
    if (argc == 2)
    	blockSize.x = atoi(argv[1]); 

	transpose(in, outDevice1, length, width, 1, blockSize);
    float err = checkCorrectFloat(outHost, outDevice1, length * width);
	printf("Error: %f\n\n", err);

    transpose(in, outDevice2, length, width, 2, blockSize);
    err = checkCorrectFloat(outHost, outDevice2, length * width);
	printf("Error: %f\n\n", err);

    transpose(in, outDevice3, length, width, 3, blockSize);
    err = checkCorrectFloat(outHost, outDevice3, length * width);
	printf("Error: %f\n\n", err);

    free(in);
    free(outHost);
    free(outDevice1);
    free(outDevice3);
    free(outDevice2);

    printf("**************************************************\n");
}
// ===================== Transpose =====================
int main(int argc, char ** argv)
{
	printDeviceInfo();
    checkTranspose(argc, argv);
    return 0;
}